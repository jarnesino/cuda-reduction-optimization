#include "sequential_cpu_reduction.cuh"

ReductionResult reduceAndMeasureTimeWithCPU(int *inputData, unsigned int size) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int sum = 0;
    for (unsigned int index = 0; index < size; index++) {
        sum += inputData[index];
    }

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{sum, elapsedTimeInMilliseconds};
}
