#include "hip/hip_runtime.h"
#include "reduce_implementations.cuh"

template<unsigned int blockSize>
__device__ void warpReduce(volatile int *data, unsigned int threadBlockIndex);

__global__ void multiple_reduce_operations_per_thread_iteration(
        int *inputData, int *outputData, unsigned int dataSize
) {
    extern __shared__ int sharedData[];

    unsigned int blockIndex = blockIdx.x;
    unsigned int threadBlockIndex = threadIdx.x;
    unsigned int elementsReducedByBlock = BLOCK_SIZE * 2;
    unsigned int index = blockIndex * elementsReducedByBlock + threadBlockIndex;
    unsigned int elementsReducedByGrid = elementsReducedByBlock * gridDim.x;
    sharedData[threadBlockIndex] = 0;
    while (index < dataSize) {
        sharedData[threadBlockIndex] += inputData[index] + inputData[index + BLOCK_SIZE];
        index += elementsReducedByGrid;
    }
    __syncthreads();

    // Do reduction in shared memory.
    if (BLOCK_SIZE >= 1024) {
        if (threadBlockIndex < 512) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 512]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 512) {
        if (threadBlockIndex < 256) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 256]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (threadBlockIndex < 128) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 128]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (threadBlockIndex < 64) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 64]; }
        __syncthreads();
    }
    if (threadBlockIndex < 32) warpReduce<BLOCK_SIZE>(sharedData, threadBlockIndex);

    // Write this block's result in shared memory.
    if (threadBlockIndex == 0) outputData[blockIndex] = sharedData[0];
}

// Template parameters are needed because device functions cannot access constants, and we want it at compile time.
template<unsigned int blockSize>
__device__ void warpReduce(volatile int *data, unsigned int threadBlockIndex) {
    if (blockSize >= 64) data[threadBlockIndex] += data[threadBlockIndex + 32];
    if (blockSize >= 32) data[threadBlockIndex] += data[threadBlockIndex + 16];
    if (blockSize >= 16) data[threadBlockIndex] += data[threadBlockIndex + 8];
    if (blockSize >= 8) data[threadBlockIndex] += data[threadBlockIndex + 4];
    if (blockSize >= 4) data[threadBlockIndex] += data[threadBlockIndex + 2];
    if (blockSize >= 2) data[threadBlockIndex] += data[threadBlockIndex + 1];
}

/*

For each sum in non-consecutive addresses, a sum operation is written in assembly (and therefore executing).
There's a type that allows us to operate over four consecutive addresses with only one operation. We can take advantage of it for performance.

*/
