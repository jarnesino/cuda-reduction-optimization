#include "hip/hip_runtime.h"
#include "reduce_implementations.cuh"
#include "../reduction.cuh"

template <unsigned int blockSize> __device__ void warpReduce(volatile int* sharedData, int threadBlockIndex);

__global__ void reduce_using_7_multiple_reduce_operations_per_thread_iteration(int *inputData, int *outputData, unsigned int dataSize) {
    extern __shared__ int sharedData[];

    unsigned int threadBlockIndex = threadIdx.x;
    unsigned int threadIndex = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = BLOCK_SIZE * 2 * gridDim.x;
    sharedData[threadBlockIndex] = 0;
    while (threadIndex < dataSize) {
        sharedData[threadBlockIndex] += inputData[threadIndex] + inputData[threadIndex + BLOCK_SIZE];
        threadIndex += gridSize;
    }
    __syncthreads();

    // Do reduction in shared memory.
    if (BLOCK_SIZE >= 1024) {
        if (threadBlockIndex < 512) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 512]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 512) {
        if (threadBlockIndex < 256) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 256]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (threadBlockIndex < 128) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 128]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (threadBlockIndex < 64) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 64]; }
        __syncthreads();
    }
    if (threadBlockIndex < 32) warpReduce<BLOCK_SIZE>(sharedData, threadBlockIndex);

    // Write this block's result in shared memory.
    if (threadBlockIndex == 0) outputData[blockIdx.x] = sharedData[0];
}

template <unsigned int blockSize>  // Needed because this is a device function which can't access the BLOCK_SIZE constant.
__device__ void warpReduce(volatile int* sharedData, int threadBlockIndex) {
    if (blockSize >= 64) sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 32];
    if (blockSize >= 32) sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 16];
    if (blockSize >= 16) sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 8];
    if (blockSize >= 8) sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 4];
    if (blockSize >= 4) sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 2];
    if (blockSize >= 2) sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 1];
}
