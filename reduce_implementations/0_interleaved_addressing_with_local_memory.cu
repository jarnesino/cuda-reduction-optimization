#include "hip/hip_runtime.h"
#include "reduce_implementations.cuh"
#include "../reduction.cuh"

__global__ void reduce_using_0_interleaved_addressing_with_local_memory(int *inputData, int *outputData, unsigned int dataSize) {
    unsigned int threadBlockIndex = threadIdx.x;
    unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();

    // Do reduction in shared memory.
    for(unsigned int amountOfElementsReduced = 1; amountOfElementsReduced < blockDim.x; amountOfElementsReduced *= 2) {
        if (threadBlockIndex % (2 * amountOfElementsReduced) == 0) {
            inputData[threadIndex] += inputData[threadIndex + amountOfElementsReduced];
        }
        __syncthreads();
    }

    // Write this block's result in shared memory.
    if (threadBlockIndex == 0) outputData[blockIdx.x] = inputData[threadIndex];
}

/*



*/
