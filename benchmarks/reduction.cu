#include "reduction.cuh"

int reduceWithKernelInDevice(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
);

TimedReductionResult reduceAndMeasureTimeWithKernel(
        const ReduceImplementationKernel &reduceKernel, int *inputData, const unsigned int dataSize
) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int value = reduceWithKernel(reduceKernel, inputData, dataSize);

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return TimedReductionResult{value, elapsedTimeInMilliseconds};
}

TimedReductionResult reduceAndMeasureTimeWithNonKernel(
        const ReduceNonKernelImplementation &implementation, int *inputData, const unsigned int dataSize
) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int value = implementation.function(inputData, dataSize);

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return TimedReductionResult{value, elapsedTimeInMilliseconds};
}
