#include "time.cuh"

TimedReductionResult reduceAndMeasureTime(
        const ReduceImplementation &reduceImplementation, int *inputData, const unsigned int dataSize
) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int value = reduceImplementation.function(inputData, dataSize);

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return TimedReductionResult{value, elapsedTimeInMilliseconds};
}
