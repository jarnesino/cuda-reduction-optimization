#include "reduction.cuh"

ReductionResult reduceAndMeasureTime(
        reduceImplementationFunction implementation,
        numberOfBlocksFunction numberOfBlocksFor,
        int *inputData,
        const unsigned int dataSize
) {

    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    const size_t dataSizeInBytes = dataSize * sizeof(int);
    unsigned int remainingElements = dataSize;
    unsigned int numberOfBlocks = numberOfBlocksFor(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc((void **) &deviceOutputData,
               numberOfBlocks * sizeof(int) * 2);  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    // Record the start event.
    hipEventRecord(startEvent, nullptr);

    // Launch kernel for each block.
    while (remainingElements > 1) {
        numberOfBlocks = numberOfBlocksFor(remainingElements);
        implementation<<<numberOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = numberOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    int value;
    hipMemcpy(&value, inputPointer, sizeof(int), hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{value, elapsedTimeInMilliseconds};
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

unsigned int numberOfBlocksForStandardReduction(const unsigned int dataSize) {
    return (dataSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
}


unsigned int numberOfBlocksForReductionWithExtraStep(const unsigned int dataSize) {
    const int blockSizedChunksReducedPerBlock = 2;
    return (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
           (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
}


unsigned int numberOfBlocksForReductionWithMultipleSteps(const unsigned int dataSize) {
    return unsignedMin(GRID_SIZE, numberOfBlocksForReductionWithExtraStep(dataSize));
}


unsigned int numberOfBlocksForReductionWithConsecutiveMemoryAddressing(const unsigned int dataSize) {
    const unsigned int blockSizedChunksReducedPerBlock = 4;
    const unsigned int blocks = (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
                                (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
    return unsignedMin(GRID_SIZE, blocks);
}

void initializeTestingDataIn(int *data, int size) {
    fillDataWith1s(data, size);
}

void fillDataWith1s(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = 1;
    }
}

unsigned int unsignedMin(unsigned int a, unsigned int b) {
    return a < b ? a : b;
}