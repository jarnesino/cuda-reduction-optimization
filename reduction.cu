#include "reduction.cuh"

int reduceWithKernel(
        const ReduceImplementationKernel &reduceKernel, int *inputData, unsigned int dataSize
);

int reduceWithKernelInDevice(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
);

ReductionResult reduceAndMeasureTime(
        const ReduceImplementationKernel &reduceKernel,
        int *inputData,
        const unsigned int dataSize
) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int value = reduceWithKernel(reduceKernel, inputData, dataSize);

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{value, elapsedTimeInMilliseconds};
}

int reduceWithKernel(
        const ReduceImplementationKernel &reduceKernel, int *inputData, unsigned int dataSize
) {
    const size_t dataSizeInBytes = dataSize * sizeof(int);
    unsigned int remainingElements = dataSize;
    unsigned int numberOfBlocks = reduceKernel.numberOfBlocksFunction(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc(
            (void **) &deviceOutputData,
            numberOfBlocks * sizeof(int) * 2
    );  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    int value = reduceWithKernelInDevice(
            reduceKernel, remainingElements, numberOfBlocks, sharedMemSize, inputPointer, outputPointer
    );

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    return value;
}

int reduceWithKernelInDevice(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        const size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
) {
    // Launch kernel for each block.
    while (remainingElements > 1) {
        numberOfBlocks = reduceImplementationKernel.numberOfBlocksFunction(remainingElements);
        reduceImplementationKernel.function<<<numberOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = numberOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    int value;
    hipMemcpy(&value, inputPointer, sizeof(int), hipMemcpyDeviceToHost);
    return value;
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}
