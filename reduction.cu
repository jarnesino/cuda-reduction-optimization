#include "reduction.cuh"

int main() {// Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    const int logDataSize = 10;
    const int dataSize = 1 << logDataSize;
    int* testingData = new int[dataSize];
    initializeTestingDataIn(testingData, dataSize);

    reduce(1, reduce_using_1_interleaved_addressing_with_divergent_branching, testingData, dataSize, startEvent, stopEvent);
    reduce(2, reduce_using_2_interleaved_addressing_with_bank_conflicts, testingData, dataSize, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}

/* Auxiliary */

void reduce(const int implementationNumber, reduceImplementationFunction implementation, int* inputData, const int dataSize, hipEvent_t startEvent, hipEvent_t stopEvent) {
    int outputData[dataSize];
    const int dataSizeInBytes = dataSize * sizeof(int);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **)&deviceInputData, dataSizeInBytes);
    hipMalloc((void **)&deviceOutputData, dataSizeInBytes);
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocks = (dataSize + threadsPerBlock - 1) / threadsPerBlock;
    size_t sharedMemSize = threadsPerBlock * sizeof(int);

    // Record the start event.
    hipEventRecord(startEvent, 0);

    // Launch kernel.
    implementation<<<blocks, threadsPerBlock, sharedMemSize>>>(deviceInputData, deviceOutputData, dataSize);

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipMemcpy(outputData, deviceOutputData, dataSizeInBytes, hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    std::cout << "*****************************************************" << std::endl;
    std::cout << "Implementation number: " << implementationNumber << std::endl;
    std::cout << "Elapsed time: " << elapsedTimeInMilliseconds << " ms" << std::endl;
    std::cout << "Reduction result: " << outputData[0] << std::endl;
    std::cout << "*****************************************************" << std::endl;

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

void initializeTestingDataIn(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = index;
    }
}
