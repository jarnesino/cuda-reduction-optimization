#include "reduction.cuh"

int reduceWithKernel(
        const ReduceImplementationKernel &reduceKernel, int *inputData, unsigned int dataSize
);

int reduceWithKernelInDevice(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
);

ReductionResult reduceAndMeasureTime(
        const ReduceImplementationKernel &reduceKernel,
        int *inputData,
        const unsigned int dataSize
) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int value = reduceWithKernel(reduceKernel, inputData, dataSize);

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{value, elapsedTimeInMilliseconds};
}
