#include "reduction.cuh"

int reduceWithCustomImplementation(const ReduceImplementationKernel &reduceImplementation, int *inputData, unsigned int dataSize);

int reduceWithKernel(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
);

ReductionResult reduceAndMeasureTime(
        const ReduceImplementationKernel &reduceImplementation,
        int *inputData,
        const unsigned int dataSize
) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    int value = reduceWithCustomImplementation(reduceImplementation, inputData, dataSize);

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{value, elapsedTimeInMilliseconds};
}

int reduceWithCustomImplementation(const ReduceImplementationKernel &reduceImplementation, int *inputData, unsigned int dataSize) {
    const size_t dataSizeInBytes = dataSize * sizeof(int);
    unsigned int remainingElements = dataSize;
    unsigned int numberOfBlocks = reduceImplementation.numberOfBlocksFunction(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc(
            (void **) &deviceOutputData,
            numberOfBlocks * sizeof(int) * 2
    );  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    int value = reduceWithKernel(
            reduceImplementation, remainingElements, numberOfBlocks, sharedMemSize, inputPointer, outputPointer
    );

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    return value;
}

int reduceWithKernel(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        const size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
) {
    // Launch kernel for each block.
    while (remainingElements > 1) {
        numberOfBlocks = reduceImplementationKernel.numberOfBlocksFunction(remainingElements);
        reduceImplementationKernel.function<<<numberOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = numberOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    int value;
    hipMemcpy(&value, inputPointer, sizeof(int), hipMemcpyDeviceToHost);
    return value;
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}
