#include "hip/hip_runtime.h"
#include "reduction.cuh"

/*

Playing around with CUDA optimizations.
https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

*/

int main() {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    const int logDataSize = 30;  // At least one element in the testing data
    const int dataSize = 1 << logDataSize;
    int *testingData = new int[dataSize];
    initializeTestingDataIn(testingData, dataSize);

    reduceAndMeasureTime(
            0, reduce_using_0_interleaved_addressing_with_local_memory,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            1, reduce_using_1_interleaved_addressing_with_divergent_branching,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            2, reduce_using_2_interleaved_addressing_with_bank_conflicts,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            3, reduce_using_3_sequential_addressing_with_idle_threads,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            4, reduce_using_4_first_add_during_load_with_loop_overhead,
            amountOfBlocksForReductionWithExtraStep, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            5, reduce_using_5_loop_unrolling_only_at_warp_level_iterations,
            amountOfBlocksForReductionWithExtraStep, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            6, reduce_using_6_complete_loop_unrolling_with_one_reduction,
            amountOfBlocksForReductionWithExtraStep, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            7, reduce_using_7_multiple_reduce_operations_per_thread_iteration,
            amountOfBlocksForReductionWithMultipleSteps, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            8, reduce_using_8_operations_for_consecutive_memory_addressing,
            amountOfBlocksForReductionWithConsecutiveMemoryAddressing, testingData, dataSize, startEvent, stopEvent
    );

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}

/* Auxiliary */

void reduceAndMeasureTime(
        const int implementationNumber,
        reduceImplementationFunction implementation,
        amountOfBlocksFunction amountOfBlocksFor,
        int *inputData,
        const int dataSize,
        hipEvent_t startEvent,
        hipEvent_t stopEvent
) {
    const size_t dataSizeInBytes = dataSize * sizeof(int);
    int remainingElements = dataSize;
    int amountOfBlocks = amountOfBlocksFor(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc((void **) &deviceOutputData,
               amountOfBlocks * sizeof(int) * 2);  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    // Record the start event.
    hipEventRecord(startEvent, 0);

    // Launch kernel for each block.
    while (remainingElements > 1) {
        amountOfBlocks = amountOfBlocksFor(remainingElements);
        implementation<<<amountOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = amountOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    int finalResult;
    hipMemcpy(&finalResult, inputPointer, sizeof(int), hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    printImplementationData(implementationNumber, elapsedTimeInMilliseconds, finalResult);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int amountOfBlocksForStandardReduction(const int dataSize) {
    return (dataSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
}


int amountOfBlocksForReductionWithExtraStep(const int dataSize) {
    const int blockSizedChunksReducedPerBlock = 2;
    return (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
           (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
}


int amountOfBlocksForReductionWithMultipleSteps(const int dataSize) {
    return min(GRID_SIZE, amountOfBlocksForReductionWithExtraStep(dataSize));
}


int amountOfBlocksForReductionWithConsecutiveMemoryAddressing(const int dataSize) {
    const int blockSizedChunksReducedPerBlock = 4;
    const int blocks = (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
                       (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
    return min(GRID_SIZE, blocks);
}

void printImplementationData(const int implementationNumber, float elapsedTimeInMilliseconds, int result) {
    printf("*** Implementation number: %d", implementationNumber);
    printf("\t Elapsed time: %f ms", elapsedTimeInMilliseconds);
    printf("\t Reduction result: %d\n", result);
}

void initializeTestingDataIn(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = rand() % 100;
    }
}
