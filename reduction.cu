#include "reduction.cuh"

/*

Playing around with CUDA optimizations.
https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

TODO: Add time complexity explanations.

*/

int main() {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // The reduction only works for up to 1024 elements (one block of threads), in order to avoid launching several kernels in different ways for each implementation.
    const int logDataSize = 10;
    const int dataSize = 1 << logDataSize;
    int* testingData = new int[dataSize];
    initializeTestingDataIn(testingData, dataSize);

    reduce(1, reduce_using_1_interleaved_addressing_with_divergent_branching, testingData, dataSize, startEvent, stopEvent);
    reduce(2, reduce_using_2_interleaved_addressing_with_bank_conflicts, testingData, dataSize, startEvent, stopEvent);
    reduce(3, reduce_using_3_sequential_addressing_with_idle_threads, testingData, dataSize, startEvent, stopEvent);
    reduce(4, reduce_using_4_first_add_during_load_with_loop_overhead, testingData, dataSize, startEvent, stopEvent);
    reduce(5, reduce_using_5_loop_unrolling_only_at_warp_level_iterations, testingData, dataSize, startEvent, stopEvent);
    reduce(6, reduce_using_6_complete_loop_unrolling_with_one_reduction, testingData, dataSize, startEvent, stopEvent);
    reduce(7, reduce_using_7_multiple_reduce_operations_per_thread_iteration, testingData, dataSize, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}

/* Auxiliary */

void reduce(const int implementationNumber, reduceImplementationFunction implementation, int* inputData, const int dataSize, hipEvent_t startEvent, hipEvent_t stopEvent) {
    int outputData[dataSize];
    const int dataSizeInBytes = dataSize * sizeof(int);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **)&deviceInputData, dataSizeInBytes);
    hipMalloc((void **)&deviceOutputData, dataSizeInBytes);
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    size_t sharedMemSize = threadsPerBlock * sizeof(int);

    // Record the start event.
    hipEventRecord(startEvent, 0);

    // Launch kernel.
    const int oneBlock = 1;
    implementation<<<oneBlock, threadsPerBlock, sharedMemSize>>>(deviceInputData, deviceOutputData, dataSize);

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipMemcpy(outputData, deviceOutputData, dataSizeInBytes, hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    printImplementationData(implementationNumber, elapsedTimeInMilliseconds, outputData[0]);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

void printImplementationData(const int implementationNumber, float elapsedTimeInMilliseconds, int result) {
    std::cout << "*** Implementation number: " << implementationNumber << "\t Elapsed time: " << elapsedTimeInMilliseconds << "\t" << "Reduction result: " << result << std::endl;
}

void initializeTestingDataIn(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = 1;
    }
}
