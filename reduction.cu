#include "reduction.cuh"

void reduceAndMeasureTime(
        const unsigned int implementationNumber,
        reduceImplementationFunction implementation,
        amountOfBlocksFunction amountOfBlocksFor,
        int *inputData,
        const unsigned int dataSize,
        hipEvent_t startEvent,
        hipEvent_t stopEvent
) {
    const size_t dataSizeInBytes = dataSize * sizeof(int);
    unsigned int remainingElements = dataSize;
    unsigned int amountOfBlocks = amountOfBlocksFor(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc((void **) &deviceOutputData,
               amountOfBlocks * sizeof(int) * 2);  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    // Record the start event.
    hipEventRecord(startEvent, nullptr);

    // Launch kernel for each block.
    while (remainingElements > 1) {
        amountOfBlocks = amountOfBlocksFor(remainingElements);
        implementation<<<amountOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = amountOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    int finalResult;
    hipMemcpy(&finalResult, inputPointer, sizeof(int), hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    printImplementationData(implementationNumber, elapsedTimeInMilliseconds, finalResult);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

unsigned int amountOfBlocksForStandardReduction(const unsigned int dataSize) {
    return (dataSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
}


unsigned int amountOfBlocksForReductionWithExtraStep(const unsigned int dataSize) {
    const int blockSizedChunksReducedPerBlock = 2;
    return (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
           (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
}


unsigned int amountOfBlocksForReductionWithMultipleSteps(const unsigned int dataSize) {
    return unsignedMin(GRID_SIZE, amountOfBlocksForReductionWithExtraStep(dataSize));
}


unsigned int amountOfBlocksForReductionWithConsecutiveMemoryAddressing(const unsigned int dataSize) {
    const unsigned int blockSizedChunksReducedPerBlock = 4;
    const unsigned int blocks = (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
                                (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
    return unsignedMin(GRID_SIZE, blocks);
}

void printImplementationData(const unsigned int implementationNumber, float elapsedTimeInMilliseconds, int result) {
    printf("*** Implementation number: %d", implementationNumber);
    printf("\t Elapsed time: %f ms", elapsedTimeInMilliseconds);
    printf("\t Reduction result: %d\n", result);
}

void initializeTestingDataIn(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = 1;
    }
}

unsigned int unsignedMin(unsigned int a, unsigned int b) {
    return a < b ? a : b;
}