#include "reduction.cuh"

ReductionResult reduceAndMeasureTime(
        ReduceImplementation reduceImplementation,
        int *inputData,
        const unsigned int dataSize
) {

    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    const size_t dataSizeInBytes = dataSize * sizeof(int);
    unsigned int remainingElements = dataSize;
    unsigned int numberOfBlocks = reduceImplementation.numberOfBlocksFunction(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc((void **) &deviceOutputData,
               numberOfBlocks * sizeof(int) * 2);  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    // Record the start event.
    hipEventRecord(startEvent, nullptr);

    // Launch kernel for each block.
    while (remainingElements > 1) {
        numberOfBlocks = reduceImplementation.numberOfBlocksFunction(remainingElements);
        reduceImplementation.function<<<numberOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = numberOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    int value;
    hipMemcpy(&value, inputPointer, sizeof(int), hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{value, elapsedTimeInMilliseconds};
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void initializeTestingDataIn(int *data, int size) {
    fillDataWith1s(data, size);
}

void fillDataWith1s(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = 1;
    }
}