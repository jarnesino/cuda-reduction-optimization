#include <iostream>
#include "reduce_implementations/reduce_implementations.cuh"
#include "reduction.h"

int main() {
    const int logDataSize = 10;
    const int dataSize = 1 << logDataSize;

    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    reduce(dataSize, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}

/* Auxiliary */

void reduce(const int dataSize, hipEvent_t startEvent, hipEvent_t stopEvent) {
    const int dataSizeInBytes = dataSize * sizeof(int);

    int inputData[dataSize];
    int outputData[dataSize];

    initializeRandomTestingDataIn(inputData, dataSize);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **)&deviceInputData, dataSizeInBytes);
    hipMalloc((void **)&deviceOutputData, dataSizeInBytes);

    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocks = (dataSize + threadsPerBlock - 1) / threadsPerBlock;
    size_t sharedMemSize = threadsPerBlock * sizeof(int);

    // Record the start event.
    hipEventRecord(startEvent, 0);

    // Launch kernel.
    reduce_using_2_interleaved_addressing_with_bank_conflicts<<<blocks, threadsPerBlock, sharedMemSize>>>(deviceInputData, deviceOutputData, dataSize);

    // Record the stop event.
    hipEventRecord(stopEvent, 0);

    // Wait for the stop event to complete.
    hipEventSynchronize(stopEvent);

    // Calculate the elapsed time in milliseconds.
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

    std::cout << "*****************************************************" << std::endl;

    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;

    hipMemcpy(outputData, deviceOutputData, dataSizeInBytes, hipMemcpyDeviceToHost);

    std::cout << "Reduction result: " << outputData[0] << std::endl;

    std::cout << "*****************************************************" << std::endl;

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

void initializeRandomTestingDataIn(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = 1;
    }
}
