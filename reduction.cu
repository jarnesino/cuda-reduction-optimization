#include "reduction.cuh"

int main() {
    const int logDataSize = 10;
    const int dataSize = 1 << logDataSize;

    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    reduce(1, reduce_using_1_interleaved_addressing_with_divergent_branching, dataSize, startEvent, stopEvent);
    reduce(2, reduce_using_2_interleaved_addressing_with_bank_conflicts, dataSize, startEvent, stopEvent);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}

/* Auxiliary */

void reduce(const int implementationNumber, reduce_implementation_function implementation, const int dataSize, hipEvent_t startEvent, hipEvent_t stopEvent) {
    const int dataSizeInBytes = dataSize * sizeof(int);

    int inputData[dataSize];
    int outputData[dataSize];
    initializeRandomTestingDataIn(inputData, dataSize);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **)&deviceInputData, dataSizeInBytes);
    hipMalloc((void **)&deviceOutputData, dataSizeInBytes);
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocks = (dataSize + threadsPerBlock - 1) / threadsPerBlock;
    size_t sharedMemSize = threadsPerBlock * sizeof(int);

    // Record the start event.
    hipEventRecord(startEvent, 0);

    // Launch kernel.
    reduce_using_2_interleaved_addressing_with_bank_conflicts<<<blocks, threadsPerBlock, sharedMemSize>>>(deviceInputData, deviceOutputData, dataSize);

    // Record the stop event and wait for it to complete.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipMemcpy(outputData, deviceOutputData, dataSizeInBytes, hipMemcpyDeviceToHost);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    std::cout << "*****************************************************" << std::endl;
    std::cout << "Elapsed time: " << elapsedTimeInMilliseconds << " ms" << std::endl;
    std::cout << "Reduction result: " << outputData[0] << std::endl;
    std::cout << "*****************************************************" << std::endl;

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

void initializeRandomTestingDataIn(int *data, int size) {
    for (int index = 0; index < size; ++index) {
        data[index] = 1;
    }
}
