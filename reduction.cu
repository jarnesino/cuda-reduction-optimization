#include <iostream>
#include <hip/hip_runtime.h>

#include "1_interleaved_addressing_with_divergent_branching.cuh"

void initializeRandomTestingData(int *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = rand() % 100;
    }
}

int main() {
    const int log_size = 19;
    const int size = 1 << log_size;
    const int bytes = size * sizeof(int);

    int h_idata[size];
    int h_odata[size];

    initializeRandomTestingData(h_idata, size);

    int *d_idata, *d_odata;
    hipMalloc((void **)&d_idata, bytes);
    hipMalloc((void **)&d_odata, bytes);

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    size_t sharedMemSize = threads * sizeof(int);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch kernel
    reduce_using_1_interleaved_addressing_with_divergent_branching<<<blocks, threads, sharedMemSize>>>(d_idata, d_odata, size);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate the elapsed time in milliseconds
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "*****************************************************" << std::endl;

    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;

    hipMemcpy(h_odata, d_odata, bytes, hipMemcpyDeviceToHost);

    std::cout << "Reduction result: " << h_odata[0] << std::endl;

    std::cout << "*****************************************************" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_idata);
    hipFree(d_odata);

    return 0;
}
