#include "thrust_reduction.cuh"

ReductionResult reduceAndMeasureTimeWithThrust(int *inputData, unsigned int size) {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // Record the CUDA start event.
    hipEventRecord(startEvent, nullptr);

    thrust::device_vector<int> deviceInputData(inputData, inputData + size);

    int sum = thrust::reduce(deviceInputData.begin(), deviceInputData.end(), 0, thrust::plus<int>());

    // Record the CUDA stop event and wait for it to complete.
    hipEventRecord(stopEvent, nullptr);
    hipEventSynchronize(stopEvent);

    float elapsedTimeInMilliseconds;
    hipEventElapsedTime(&elapsedTimeInMilliseconds, startEvent, stopEvent);

    // Destroy the CUDA events for timing.
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return ReductionResult{sum, elapsedTimeInMilliseconds};
}
