#include "reduction.cuh"

/*

Playing around with CUDA optimizations.
https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

*/

int main() {
    // Create CUDA events for timing.
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    const unsigned int logDataSize = 30;  // At least one element in the testing data
    const unsigned int dataSize = 1 << logDataSize;
    int *testingData = new int[dataSize];
    initializeTestingDataIn(testingData, dataSize);

    reduceAndMeasureTime(
            0, reduce_using_0_interleaved_addressing_with_local_memory,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            1, reduce_using_1_interleaved_addressing_with_divergent_branching,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            2, reduce_using_2_interleaved_addressing_with_bank_conflicts,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            3, reduce_using_3_sequential_addressing_with_idle_threads,
            amountOfBlocksForStandardReduction, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            4, reduce_using_4_first_add_during_load_with_loop_overhead,
            amountOfBlocksForReductionWithExtraStep, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            5, reduce_using_5_loop_unrolling_only_at_warp_level_iterations,
            amountOfBlocksForReductionWithExtraStep, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            6, reduce_using_6_complete_loop_unrolling_with_one_reduction,
            amountOfBlocksForReductionWithExtraStep, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            7, reduce_using_7_multiple_reduce_operations_per_thread_iteration,
            amountOfBlocksForReductionWithMultipleSteps, testingData, dataSize, startEvent, stopEvent
    );
    reduceAndMeasureTime(
            8, reduce_using_8_operations_for_consecutive_memory_addressing,
            amountOfBlocksForReductionWithConsecutiveMemoryAddressing, testingData, dataSize, startEvent, stopEvent
    );

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return EXIT_SUCCESS;
}