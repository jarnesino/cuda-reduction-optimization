#include "hip/hip_runtime.h"
#include "reduce_kernels.cuh"

__global__ void sequentialAddressingWithLocalMemoryAndIdleThreads(
        int *inputData, int *outputData, unsigned int dataSize
) {
    unsigned int blockSize = blockDim.x;
    unsigned int blockIndex = blockIdx.x;
    unsigned int threadBlockIndex = threadIdx.x;
    unsigned int threadIndex = blockIndex * blockSize + threadBlockIndex;
    __syncthreads();

    // Do reduction.
    for (
            unsigned int numberOfElementsToReduce = blockSize >> 1;
            numberOfElementsToReduce > 0;
            numberOfElementsToReduce >>= 1
            ) {
        if (threadBlockIndex <
            numberOfElementsToReduce) {  // This if statement makes many threads idle threads in each iteration.
            inputData[threadIndex] += inputData[threadIndex + numberOfElementsToReduce];
        }
        __syncthreads();
    }

    // Write this block's result.
    if (threadBlockIndex == 0) outputData[blockIndex] = inputData[threadIndex];
}

int reduceWithSequentialAddressingWithLocalMemoryAndIdleThreads(int *data, unsigned int dataSize) {
    ReduceImplementationKernel kernel = {sequentialAddressingWithLocalMemoryAndIdleThreads, numberOfBlocksForStandardReduction};
    return reduceWithKernel(kernel, data, dataSize);
}
