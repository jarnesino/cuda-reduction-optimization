#include "reduce_kernels.cuh"

unsigned int numberOfBlocksForStandardReduction(const unsigned int dataSize) {
    return (dataSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
}


unsigned int numberOfBlocksForReductionWithExtraStep(const unsigned int dataSize) {
    const int blockSizedChunksReducedPerBlock = 2;
    return (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
           (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
}


unsigned int numberOfBlocksForReductionWithMultipleSteps(const unsigned int dataSize) {
    return unsignedMin(GRID_SIZE, numberOfBlocksForReductionWithExtraStep(dataSize));
}


unsigned int numberOfBlocksForReductionWithConsecutiveMemoryAddressing(const unsigned int dataSize) {
    const unsigned int blockSizedChunksReducedPerBlock = 4;
    const unsigned int blocks = (dataSize + BLOCK_SIZE * blockSizedChunksReducedPerBlock - 1) /
                                (BLOCK_SIZE * blockSizedChunksReducedPerBlock);
    return unsignedMin(GRID_SIZE, blocks);
}

unsigned int unsignedMin(unsigned int a, unsigned int b) {
    return a < b ? a : b;
}

void checkForCUDAErrors() {
    hipError_t result = hipGetLastError();
    if (result != hipSuccess) {
        std::cerr << "CUDA error: ";
        std::cerr << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int reduceWithKernelInDevice(
        const ReduceImplementationKernel &reduceImplementationKernel,
        unsigned int remainingElements,
        unsigned int numberOfBlocks,
        const size_t sharedMemSize,
        int *inputPointer,
        int *outputPointer
) {
    // Launch kernel for each block.
    while (remainingElements > 1) {
        numberOfBlocks = reduceImplementationKernel.numberOfBlocksFunction(remainingElements);
        reduceImplementationKernel.function<<<numberOfBlocks, BLOCK_SIZE, sharedMemSize>>>(
                inputPointer, outputPointer, remainingElements
        );
        hipDeviceSynchronize();
        checkForCUDAErrors();

        remainingElements = numberOfBlocks;
        inputPointer = outputPointer;
        outputPointer += remainingElements;
    }

    int value;
    hipMemcpy(&value, inputPointer, sizeof(int), hipMemcpyDeviceToHost);
    return value;
}

int reduceWithKernel(
        const ReduceImplementationKernel &reduceKernel, int *inputData, const unsigned int dataSize
) {
    const size_t dataSizeInBytes = dataSize * sizeof(int);
    unsigned int remainingElements = dataSize;
    unsigned int numberOfBlocks = reduceKernel.numberOfBlocksFunction(remainingElements);

    int *deviceInputData, *deviceOutputData;
    hipMalloc((void **) &deviceInputData, dataSizeInBytes);
    hipMalloc(
            (void **) &deviceOutputData,
            numberOfBlocks * sizeof(int) * 2
    );  // Allocate double the memory for use in subsequent layers.
    hipMemcpy(deviceInputData, inputData, dataSizeInBytes, hipMemcpyHostToDevice);
    const size_t sharedMemSize = BLOCK_SIZE * sizeof(int);

    int *inputPointer = deviceInputData;
    int *outputPointer = deviceOutputData;

    int value = reduceWithKernelInDevice(
            reduceKernel, remainingElements, numberOfBlocks, sharedMemSize, inputPointer, outputPointer
    );

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    return value;
}
