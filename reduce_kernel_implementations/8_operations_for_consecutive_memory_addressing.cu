#include "hip/hip_runtime.h"
#include "reduce_kernels.cuh"

// Template parameters are needed because device functions cannot access constants, and we want it at compile time.
template<unsigned int blockSize>
__device__ void warpReduce(volatile int *data, unsigned int threadBlockIndex) {
    if (blockSize >= 64) data[threadBlockIndex] += data[threadBlockIndex + 32];
    if (blockSize >= 32) data[threadBlockIndex] += data[threadBlockIndex + 16];
    if (blockSize >= 16) data[threadBlockIndex] += data[threadBlockIndex + 8];
    if (blockSize >= 8) data[threadBlockIndex] += data[threadBlockIndex + 4];
    if (blockSize >= 4) data[threadBlockIndex] += data[threadBlockIndex + 2];
    if (blockSize >= 2) data[threadBlockIndex] += data[threadBlockIndex + 1];
}

__global__ void operations_for_consecutive_memory_addressing(
        int *inputData, int *outputData, unsigned int dataSize
) {
    extern __shared__ int sharedData[];

    unsigned int blockIndex = blockIdx.x;
    unsigned int threadBlockIndex = threadIdx.x;
    int4 *inputDataForConsecutiveAccessing = (int4 *) inputData;
    unsigned int elementsReducedByBlock = BLOCK_SIZE;
    unsigned int index = blockIndex * elementsReducedByBlock + threadBlockIndex;
    unsigned int elementsReducedByGrid = elementsReducedByBlock * gridDim.x;
    sharedData[threadBlockIndex] = 0;
    while (index < (dataSize >> 2)) {
        int4 input = inputDataForConsecutiveAccessing[index];
        sharedData[threadBlockIndex] += input.x + input.y + input.z + input.w;
        index += elementsReducedByGrid;
    }
    __syncthreads();

    // Do reduction in shared memory.
    if (BLOCK_SIZE >= 1024) {
        if (threadBlockIndex < 512) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 512]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 512) {
        if (threadBlockIndex < 256) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 256]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (threadBlockIndex < 128) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 128]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (threadBlockIndex < 64) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 64]; }
        __syncthreads();
    }
    if (threadBlockIndex < 32) warpReduce<BLOCK_SIZE>(sharedData, threadBlockIndex);

    // Write this block's result.
    if (threadBlockIndex == 0) outputData[blockIndex] = sharedData[0];
}

int reduceWithOperationsForConsecutiveMemoryAddressing(int *data, unsigned int dataSize) {
    ReduceImplementationKernel kernel = {
            operations_for_consecutive_memory_addressing, numberOfBlocksForReductionWithConsecutiveMemoryAddressing
    };
    return reduceWithKernel(kernel, data, dataSize);
}

/*

This operates over four consecutive memory addresses with operations that are optimized for such usage.
However, memory is still slower than registers. What if we could access other thread's registers instead of shared
memory?

*/
