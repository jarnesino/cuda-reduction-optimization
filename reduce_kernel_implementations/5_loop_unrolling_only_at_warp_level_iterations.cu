#include "hip/hip_runtime.h"
#include "reduce_kernels.cuh"

__device__ void warpReduce(volatile int *data, const unsigned int threadBlockIndex) {
    data[threadBlockIndex] += data[threadBlockIndex + 32];
    data[threadBlockIndex] += data[threadBlockIndex + 16];
    data[threadBlockIndex] += data[threadBlockIndex + 8];
    data[threadBlockIndex] += data[threadBlockIndex + 4];
    data[threadBlockIndex] += data[threadBlockIndex + 2];
    data[threadBlockIndex] += data[threadBlockIndex + 1];
}

__global__ void loopUnrollingOnlyAtWarpLevelIterations(
        int *inputData, int *outputData, unsigned int dataSize
) {
    extern __shared__ int sharedData[];

    unsigned int blockSize = blockDim.x;
    unsigned int blockIndex = blockIdx.x;
    unsigned int threadBlockIndex = threadIdx.x;
    unsigned int threadIndex = blockIndex * blockSize * 2 + threadBlockIndex;
    sharedData[threadBlockIndex] = inputData[threadIndex] + inputData[threadIndex + blockSize];
    __syncthreads();

    // Do reduction in shared memory.
    for (
            unsigned int numberOfElementsToReduce = blockSize >> 1;
            numberOfElementsToReduce > 32;
            numberOfElementsToReduce >>= 1
            ) {  // This loop produces instruction overhead.
        if (threadBlockIndex < numberOfElementsToReduce) {
            sharedData[threadBlockIndex] += sharedData[threadBlockIndex + numberOfElementsToReduce];
        }
        __syncthreads();
    }
    if (threadBlockIndex < 32) warpReduce(sharedData, threadBlockIndex);

    // Write this block's result.
    if (threadBlockIndex == 0) outputData[blockIndex] = sharedData[0];
}

int reduceWithLoopUnrollingOnlyAtWarpLevelIterations(int *data, unsigned int dataSize) {
    ReduceImplementationKernel kernel = {
            loopUnrollingOnlyAtWarpLevelIterations, numberOfBlocksForReductionWithExtraStep
    };
    return reduceWithKernel(kernel, data, dataSize);
}

/*

Loops like this one produce instruction overhead.

Completely unrolling the loop could be a good solution.
We could know what the limitation for threads per block is.
In this case, it's 1024 (2^10).
We can use this to completely unroll the loop in the kernel.
Given that we don't know the block size at compile time, we can use C++ template parameters, supported by CUDA in host and device functions.

*/
