#include "hip/hip_runtime.h"
#include "reduce_kernels.cuh"

__inline__ __device__ int warpReduce(int val) {
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);  // Shuffle from the other thread's sum variable register.
    return val;
}

__global__ void shuffle_down(
        int *inputData, int *outputData, unsigned int dataSize
) {
    extern __shared__ int sharedData[];

    unsigned int blockIndex = blockIdx.x;
    unsigned int threadBlockIndex = threadIdx.x;
    int4 *inputDataForConsecutiveAccessing = (int4 *) inputData;
    unsigned int elementsReducedByBlock = BLOCK_SIZE;
    unsigned int index = blockIndex * elementsReducedByBlock + threadBlockIndex;
    unsigned int elementsReducedByGrid = elementsReducedByBlock * gridDim.x;
    sharedData[threadBlockIndex] = 0;
    while (index < (dataSize >> 2)) {
        int4 input = inputDataForConsecutiveAccessing[index];
        sharedData[threadBlockIndex] += input.x + input.y + input.z + input.w;
        index += elementsReducedByGrid;
    }
    dataSize = dataSize >> 2;
    __syncthreads();

    // Do reduction in shared memory.
    if (BLOCK_SIZE >= 1024) {
        if (threadBlockIndex < 512) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 512]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 512) {
        if (threadBlockIndex < 256) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 256]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256) {
        if (threadBlockIndex < 128) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 128]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128) {
        if (threadBlockIndex < 64) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 64]; }
        __syncthreads();
    }
    if (BLOCK_SIZE >= 64) {
        if (threadBlockIndex < 32) { sharedData[threadBlockIndex] += sharedData[threadBlockIndex + 32]; }
        __syncthreads();
    }

    if (threadBlockIndex < 32) {
        int sum = sharedData[threadBlockIndex];
        sum = warpReduce(sum);  // Reduce last warp
        if (threadBlockIndex == 0) outputData[blockIndex] = sum;  // Write this block's result.
    }
}

int reduceWithShuffleDown(int *data, unsigned int dataSize) {
    ReduceImplementationKernel kernel = {shuffle_down, numberOfBlocksForReductionWithConsecutiveMemoryAddressing};
    return reduceWithKernel(kernel, data, dataSize);
}

/*

The shuffle operations take advantage of threads between a warp being able to share registers.
Based on:
    https://developer.nvidia.com/blog/faster-parallel-reductions-kepler/
    https://github.com/NVIDIA-developer-blog/code-samples/tree/52b16fac9a135ca12b6c4d53529128d2672cc6ad/posts/parallel_reduction_with_shfl

*/
