#include "hip/hip_runtime.h"
#include "reduce_kernels.cuh"

__global__ void interleavedAddressingWithGlobalMemoryAndBankConflicts(
        int *inputData, int *outputData, unsigned int dataSize
) {
    unsigned int blockSize = blockDim.x;
    unsigned int blockIndex = blockIdx.x;
    unsigned int threadBlockIndex = threadIdx.x;
    unsigned int threadIndex = blockIndex * blockSize + threadBlockIndex;

    // Do reduction in global memory. Causes slow access speeds.
    for (unsigned int numberOfElementsReduced = 1; numberOfElementsReduced < blockSize; numberOfElementsReduced <<= 1) {
        unsigned int index = (numberOfElementsReduced * threadBlockIndex) << 1;
        if (index < blockSize) {
            inputData[blockSize * blockIndex + index] += inputData[blockSize * blockIndex + index + numberOfElementsReduced];
        }
        __syncthreads();
    }

    // Write this block's result.
    if (threadBlockIndex == 0) outputData[blockIndex] = inputData[threadIndex];
}

int reduceWithInterleavedAddressingWithGlobalMemoryAndBankConflicts(int *data, unsigned int dataSize) {
    ReduceImplementationKernel kernel = {
            interleavedAddressingWithGlobalMemoryAndBankConflicts, numberOfBlocksForStandardReduction
    };
    return reduceWithKernel(kernel, data, dataSize);
}
